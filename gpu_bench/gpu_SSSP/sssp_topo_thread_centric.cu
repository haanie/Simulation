#include "hip/hip_runtime.h"
//=================================================================//
// CUDA SSSP kernel
// Topological-Driven: one node per thread, thread_centric,
//      use atomicMin
// Reference: 
// Sungpack Hong, et al. Accelerating CUDA graph algorithms 
//      at maximum warp
//=================================================================//
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>

#include "cudaGraph.h"

__global__ void initialize(uint32_t * d_vpl, uint32_t * d_update, bool * d_mask, uint64_t num_vertex)
{
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if ( tid < num_vertex )
    {
        d_vpl[tid] = MY_INFINITY;
        d_update[tid] = MY_INFINITY;
        d_mask[tid] = false;
    }
}

__global__
void kernel(uint32_t * vplist, 
        uint32_t * eplist, 
        uint32_t * update, 
        bool * mask, 
        cudaGraph graph) 
{
	uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= graph.vertex_cnt) return;
    if (mask[tid])
    {
        uint64_t start, end;
        start = graph.get_firstedge_index(tid);
        end = graph.get_edge_index_end(tid);
        
        uint32_t cost = vplist[tid];

        for (uint64_t i=start; i<end; i++)
        {
            uint64_t vid = graph.get_edge_dest(i);
            atomicMin(&(update[vid]), cost+eplist[i]);
        }
        mask[tid] = false;
    }
}
__global__
void kernel2(uint32_t * vplist, 
        uint32_t * update, 
        bool * mask, 
        cudaGraph graph, 
        bool *changed) 
{
	uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= graph.vertex_cnt) return;

    if (vplist[tid] > update[tid])
    {
        vplist[tid] = update[tid];
        mask[tid] = true;
        *changed = true;
    }
    else
    {
        update[tid] = vplist[tid];
    }
}


void cuda_SSSP(uint64_t * vertexlist, 
        uint64_t * edgelist, 
        uint32_t * vproplist,
        uint32_t * eproplist,
        uint64_t vertex_cnt, 
        uint64_t edge_cnt,
        uint64_t root)
{
    uint32_t * device_vpl = 0;
    uint32_t * device_epl = 0;
    uint32_t * device_update = 0;
    bool * device_mask = 0;
    bool * device_over = 0;

    float h2d_copy_time = 0; // host to device data transfer time
    float d2h_copy_time = 0; // device to host data transfer time
    float kernel_time = 0;   // kernel execution time

    int device;
    hipGetDevice(&device);
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp,device);


    // Try to use as many threads as possible so that each thread
    //      is processing one vertex. If max thread is reached, 
    //      split them into multiple blocks.
    unsigned int num_thread_per_block = (unsigned int) vertex_cnt;
    if (num_thread_per_block > devProp.maxThreadsPerBlock)
        num_thread_per_block = devProp.maxThreadsPerBlock;
    unsigned int num_block = (unsigned int)ceil( vertex_cnt/(double)num_thread_per_block );

    // malloc of gpu side
    cudaErrCheck( hipMalloc((void**)&device_vpl, vertex_cnt*sizeof(uint32_t)) );
    cudaErrCheck( hipMalloc((void**)&device_mask, vertex_cnt*sizeof(bool)) );
    cudaErrCheck( hipMalloc((void**)&device_update, vertex_cnt*sizeof(uint32_t)) );
    cudaErrCheck( hipMalloc((void**)&device_epl, edge_cnt*sizeof(uint32_t)) );
    cudaErrCheck( hipMalloc((void**)&device_over, sizeof(bool)) );

    hipEvent_t start_event, stop_event;
    cudaErrCheck( hipEventCreate(&start_event) );
    cudaErrCheck( hipEventCreate(&stop_event) );
    
    // initialization
    initialize<<<num_block, num_thread_per_block>>>(device_vpl, device_update, device_mask, vertex_cnt);
    
    // prepare graph struct
    //  one for host side, one for device side
    cudaGraph h_graph, d_graph;
    // here copy only the pointers
    h_graph.read(vertexlist, edgelist, vertex_cnt, edge_cnt);

    uint32_t zeronum=0;
    bool truenum=true;
    // memcpy from host to device
    hipEventRecord(start_event, 0);
   
    // copy graph data to device
    h_graph.cudaGraphCopy(&d_graph);
    // set root vprop
    cudaErrCheck( hipMemcpy(&(device_vpl[root]), &zeronum, sizeof(uint32_t), 
                hipMemcpyHostToDevice) );
    cudaErrCheck( hipMemcpy(&(device_mask[root]), &truenum, sizeof(bool), 
                hipMemcpyHostToDevice) );
    // copy edge prop to device
    cudaErrCheck( hipMemcpy(device_epl, eproplist, edge_cnt*sizeof(uint32_t), 
                hipMemcpyHostToDevice) );

    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&h2d_copy_time, start_event, stop_event);

    
    // BFS traversal
    bool stop;
    hipEventRecord(start_event, 0);
   
    int curr=0; 
    do
    {
        // Each iteration processes 
        //      one level of BFS traversal
        stop = false;
        cudaErrCheck( hipMemcpy(device_over, &stop, sizeof(bool), hipMemcpyHostToDevice) );

        kernel<<<num_block, num_thread_per_block>>>(device_vpl, device_epl,
                device_update, device_mask, d_graph);
        kernel2<<<num_block, num_thread_per_block>>>(device_vpl, device_update, 
                device_mask, d_graph, device_over);


        cudaErrCheck( hipMemcpy(&stop, device_over, sizeof(bool), hipMemcpyDeviceToHost) );

        curr++;
    }while(stop);

    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&kernel_time, start_event, stop_event);


    hipEventRecord(start_event, 0);

    cudaErrCheck( hipMemcpy(vproplist, device_vpl, vertex_cnt*sizeof(uint32_t), 
                hipMemcpyDeviceToHost) );
    
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&d2h_copy_time, start_event, stop_event);

    printf("== iteration #: %d\n", curr);
#ifndef ENABLE_VERIFY
    printf("== host->device copy time: %f ms\n", h2d_copy_time);
    printf("== device->host copy time: %f ms\n", d2h_copy_time);
    printf("== kernel time: %f ms\n", kernel_time);
#endif
    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);

    // free graph struct on device side
    d_graph.cudaGraphFree();

    cudaErrCheck( hipFree(device_vpl) );
    cudaErrCheck( hipFree(device_epl) );
    cudaErrCheck( hipFree(device_update) );
    cudaErrCheck( hipFree(device_mask) );
}

