#include "hip/hip_runtime.h"
//=================================================================//
// CUDA SSSP kernel
// Data-Driven: one edge per thread, warp_centric,
//      use atomicMin & atomicAdd
// Reference: 
// Sungpack Hong, et al. Accelerating CUDA graph algorithms 
//      at maximum warp
//=================================================================//
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>

#include "cudaGraph.h"

#define WORKLIST_SIZE   8777216
#define LOCAL_SIZE      128

#define WARP_SZ     32
#define CHUNK_SZ    32

// a dummy worklist that you can only push or clear
typedef struct my_worklist
{
    void init(void)
    {
        cudaErrCheck( hipMalloc((void**)&item_array, WORKLIST_SIZE*sizeof(uint64_t)) );
        cudaErrCheck( hipMalloc((void**)&end, sizeof(uint32_t)) );
        clear();
    }

    void clear(void)
    {
        uint32_t zeronum=0;
        cudaErrCheck( hipMemcpy(end, &zeronum, sizeof(uint32_t), 
                hipMemcpyHostToDevice) );
    }

    void free(void)
    {
        cudaErrCheck( hipFree(item_array) );
        cudaErrCheck( hipFree(end) );
    }
    __device__ void pushRange(uint64_t * from_array, uint32_t num)
    {
        uint32_t old_end = atomicAdd(end, num);
        for (uint32_t i=0;i<num;i++)
        {
            item_array[i+old_end] = from_array[i];
        }
    }
    __device__ inline uint64_t get_item(unsigned index)
    {
        return item_array[index];
    }
    __device__ inline uint32_t get_item_num(void)
    {
            return (*end);
    }
    void host_initPush(uint64_t * from_array, uint32_t num)
    {
        cudaErrCheck( hipMemcpy(end, &num, sizeof(uint32_t), 
                hipMemcpyHostToDevice) );
        cudaErrCheck( hipMemcpy(item_array, from_array, num*sizeof(uint64_t), 
                hipMemcpyHostToDevice) );
    }

    uint64_t *item_array;
    uint32_t *end;
}my_worklist;


__global__ void initialize(uint32_t * d_vpl, uint32_t * d_update, bool * d_mask, uint64_t num_vertex)
{
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if ( tid < num_vertex )
    {
        d_vpl[tid] = MY_INFINITY;
        d_update[tid] = MY_INFINITY;
    }
}

__global__
void kernel(uint32_t * vplist, 
        uint32_t * eplist, 
        uint32_t * update,  
        cudaGraph graph,
        my_worklist inworklist, 
        my_worklist outworklist) 
{
	uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t lane_id = tid % WARP_SZ;
    unsigned warp_id = tid / WARP_SZ;

    unsigned task_start = warp_id * CHUNK_SZ;
    unsigned task_end = task_start + CHUNK_SZ;

    if (task_start >= inworklist.get_item_num()) return;
    if (task_end > inworklist.get_item_num()) 
        task_end = inworklist.get_item_num();
    
    uint64_t local_worklist[LOCAL_SIZE]; 
    uint32_t work_size=0;
    
    for (unsigned id=task_start; id<task_end; id++)
    {
        uint64_t v = inworklist.get_item(id);
        uint32_t cost = vplist[v];
        uint64_t edge_ptr = graph.get_firstedge_index(v);
        uint64_t num_edge = graph.get_edge_index_end(v) - edge_ptr;

        for (uint64_t i=lane_id;i<num_edge;i+=WARP_SZ)
        {
            uint64_t vid = graph.get_edge_dest(i+edge_ptr);
            uint32_t new_dist = cost + eplist[i+edge_ptr];
            if ( update[vid] > new_dist)
            {
                if (atomicMin(&(update[vid]), new_dist)>new_dist)
                    local_worklist[work_size++]=vid;
                if (work_size==LOCAL_SIZE)
                {     
                    outworklist.pushRange(local_worklist, work_size);
                    work_size = 0;
                }
            }
        }

    }

    // push local worklist to shared worklist
    outworklist.pushRange(local_worklist, work_size);
}
__global__
void kernel2(uint32_t * vplist, 
        uint32_t * update,
        cudaGraph graph, 
        my_worklist inworklist) 
{
	uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= inworklist.get_item_num()) return;

    uint64_t v = inworklist.get_item(tid);

    if (vplist[v] > update[v])
    {
        vplist[v] = update[v];
    }
}


void cuda_SSSP(uint64_t * vertexlist, 
        uint64_t * edgelist, 
        uint32_t * vproplist,
        uint32_t * eproplist,
        uint64_t vertex_cnt, 
        uint64_t edge_cnt,
        uint64_t root)
{
    uint32_t * device_vpl = 0;
    uint32_t * device_epl = 0;
    uint32_t * device_update = 0;
    bool * device_mask = 0;
    bool * device_over = 0;

    float h2d_copy_time = 0; // host to device data transfer time
    float d2h_copy_time = 0; // device to host data transfer time
    float kernel_time = 0;   // kernel execution time

    int device;
    hipGetDevice(&device);
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp,device);


    // Try to use as many threads as possible so that each thread
    //      is processing one vertex. If max thread is reached, 
    //      split them into multiple blocks.
    unsigned int num_thread_per_block = (unsigned int) vertex_cnt;
    if (num_thread_per_block > devProp.maxThreadsPerBlock)
        num_thread_per_block = devProp.maxThreadsPerBlock;
    unsigned int num_block = (unsigned int)ceil( vertex_cnt/(double)num_thread_per_block );

    // malloc of gpu side
    cudaErrCheck( hipMalloc((void**)&device_vpl, vertex_cnt*sizeof(uint32_t)) );
    cudaErrCheck( hipMalloc((void**)&device_update, vertex_cnt*sizeof(uint32_t)) );
    cudaErrCheck( hipMalloc((void**)&device_epl, edge_cnt*sizeof(uint32_t)) );
    cudaErrCheck( hipMalloc((void**)&device_over, sizeof(bool)) );

    hipEvent_t start_event, stop_event;
    cudaErrCheck( hipEventCreate(&start_event) );
    cudaErrCheck( hipEventCreate(&stop_event) );
    
    // initialization
    initialize<<<num_block, num_thread_per_block>>>(device_vpl, device_update, device_mask, vertex_cnt);
    
    // prepare graph struct
    //  one for host side, one for device side
    cudaGraph h_graph, d_graph;
    // here copy only the pointers
    h_graph.read(vertexlist, edgelist, vertex_cnt, edge_cnt);

    // initialize the worklists for in & out
    my_worklist worklist1, worklist2;
    worklist1.init();
    worklist2.init();

    my_worklist * in_worklist = &worklist1;
    my_worklist * out_worklist = &worklist2;

    in_worklist->host_initPush(&root, 1);

    uint32_t zeronum=0;
    // memcpy from host to device
    hipEventRecord(start_event, 0);
   
    // copy graph data to device
    h_graph.cudaGraphCopy(&d_graph);
    // set root vprop
    cudaErrCheck( hipMemcpy(&(device_vpl[root]), &zeronum, sizeof(uint32_t), 
                hipMemcpyHostToDevice) );
    // copy edge prop to device
    cudaErrCheck( hipMemcpy(device_epl, eproplist, edge_cnt*sizeof(uint32_t), 
                hipMemcpyHostToDevice) );

    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&h2d_copy_time, start_event, stop_event);

    
    // traversal
    hipEventRecord(start_event, 0);
   
    int curr=0;  
    unsigned wl_size=1;
    num_block = 1;
    num_thread_per_block = 1; 
    unsigned num_block_chunked = 1;
    
    while(wl_size!=0)
    {
        kernel<<<num_block_chunked, num_thread_per_block>>>(device_vpl, device_epl,
                device_update, d_graph, *in_worklist, *out_worklist);
        
        my_worklist * temp=in_worklist;
        in_worklist = out_worklist;
        out_worklist = temp;

        cudaErrCheck( hipMemcpy(&wl_size, in_worklist->end, sizeof(uint32_t), hipMemcpyDeviceToHost) );
        out_worklist->clear();
        
        num_thread_per_block = (unsigned int) wl_size;
        if (num_thread_per_block > devProp.maxThreadsPerBlock)
            num_thread_per_block = devProp.maxThreadsPerBlock;
        num_block = (unsigned int)ceil( wl_size/(double)num_thread_per_block );
        num_block_chunked = (unsigned int)ceil( num_block/(double)CHUNK_SZ )*WARP_SZ;
        
        kernel2<<<num_block, num_thread_per_block>>>(device_vpl, device_update, 
                d_graph, *in_worklist);

        curr++;
    }

    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&kernel_time, start_event, stop_event);


    hipEventRecord(start_event, 0);

    cudaErrCheck( hipMemcpy(vproplist, device_vpl, vertex_cnt*sizeof(uint32_t), 
                hipMemcpyDeviceToHost) );
    
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&d2h_copy_time, start_event, stop_event);

    printf("== iteration #: %d\n", curr);
#ifndef ENABLE_VERIFY
    printf("== host->device copy time: %f ms\n", h2d_copy_time);
    printf("== device->host copy time: %f ms\n", d2h_copy_time);
    printf("== kernel time: %f ms\n", kernel_time);
#endif
    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);

    // free graph struct on device side
    d_graph.cudaGraphFree();

    in_worklist->free();
    out_worklist->free();

    cudaErrCheck( hipFree(device_vpl) );
    cudaErrCheck( hipFree(device_epl) );
    cudaErrCheck( hipFree(device_update) );
    cudaErrCheck( hipFree(device_mask) );
}

