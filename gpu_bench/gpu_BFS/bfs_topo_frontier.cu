#include "hip/hip_runtime.h"
//=================================================================//
// CUDA BFS kernel
// Topological-Driven: one node per thread, no atomic instructions
// Reference: 
//      Pawan Harish, Accelerating large graph algorithms 
//                  on the GPU using CUDA (HiPC 2007)
//=================================================================//
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>

#include "cudaGraph.h"

__global__ void initialize(bool * d_graph_frontier,
                        bool * d_updating_graph_frontier,
                        bool * d_graph_visited,
                        uint32_t * d_graph_property,
                        uint64_t num_vertex)
{
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if ( tid < num_vertex )
    {
        d_graph_frontier[tid] = false;
        d_updating_graph_frontier[tid] = false;
        d_graph_visited[tid] = false;
        d_graph_property[tid] = MY_INFINITY;
    }
}


__global__ void BFS_kernel_1(
        cudaGraph d_graph,
        bool * device_graph_frontier, 
        bool * device_updating_graph_frontier, 
        bool * device_graph_visited, 
        uint32_t * device_vpl 
        )
{
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
   
    if ( tid<d_graph.vertex_cnt && device_graph_frontier[tid] )
    {
        device_graph_frontier[tid] = false;
        uint64_t eidx = d_graph.get_firstedge_index(tid);
        uint64_t eidx_end = d_graph.get_edge_index_end(tid);

        for (size_t i=eidx; i<eidx_end; i++)
        {
            uint64_t vid = d_graph.get_edge_dest(i);
            if (device_graph_visited[vid]==false)
            {
                device_vpl[vid] = device_vpl[tid]+1;
                device_updating_graph_frontier[vid] = true;
            }
        }
    }
}

__global__ void BFS_kernel_2(
        bool * device_graph_frontier, 
        bool * device_updating_graph_frontier, 
        bool * device_graph_visited, 
        bool * device_over, 
        uint64_t vl_sz 
        )
{
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if ( tid < vl_sz && device_updating_graph_frontier[tid] )
    {
        device_graph_frontier[tid] = true;
        device_graph_visited[tid] = true;
        device_updating_graph_frontier[tid] = false;
        *device_over = true;
    } 
}


void cuda_BFS(uint64_t * vertexlist, 
        uint64_t * edgelist, uint32_t * vproplist,
        uint64_t vertex_cnt, uint64_t edge_cnt,
        uint64_t root)
{
    uint32_t * device_vpl = 0;
    bool * device_graph_frontier = 0;
    bool * device_updating_graph_frontier = 0;
    bool * device_graph_visited = 0;
    bool * device_over = 0;

    float h2d_copy_time = 0; // host to device data transfer time
    float d2h_copy_time = 0; // device to host data transfer time
    float kernel_time = 0;   // kernel execution time
    
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp,device);


    // Try to use as many threads as possible so that each thread
    //      is processing one vertex. If max thread is reached, 
    //      split them into multiple blocks.
    unsigned int num_thread_per_block = (unsigned int) vertex_cnt;
    if (num_thread_per_block > devProp.maxThreadsPerBlock)
        num_thread_per_block = devProp.maxThreadsPerBlock;
    unsigned int num_block = (unsigned int)ceil( vertex_cnt/(double)num_thread_per_block );

    // malloc of gpu side
    cudaErrCheck( hipMalloc((void**)&device_vpl, vertex_cnt*sizeof(uint32_t)) );

    cudaErrCheck( hipMalloc((void**)&device_graph_frontier, vertex_cnt*sizeof(bool)) );
    cudaErrCheck( hipMalloc((void**)&device_updating_graph_frontier, vertex_cnt*sizeof(bool)) );
    cudaErrCheck( hipMalloc((void**)&device_graph_visited, vertex_cnt*sizeof(bool)) );

    cudaErrCheck( hipMalloc((void**)&device_over, sizeof(bool)) );

    hipEvent_t start_event, stop_event;
    cudaErrCheck( hipEventCreate(&start_event) );
    cudaErrCheck( hipEventCreate(&stop_event) );
    
    // initialization
    initialize<<<num_block, num_thread_per_block>>>( device_graph_frontier,
                        device_updating_graph_frontier,
                        device_graph_visited,
                        device_vpl,
                        vertex_cnt);
    
    // prepare graph struct
    //  one for host side, one for device side
    cudaGraph h_graph, d_graph;
    // here copy only the pointers
    h_graph.read(vertexlist, edgelist, vertex_cnt, edge_cnt);

    bool true_flag=true;
    uint32_t zero_flag=0;
    // memcpy from host to device
    hipEventRecord(start_event, 0);
   
    // copy graph data to device
    h_graph.cudaGraphCopy(&d_graph);

    cudaErrCheck( hipMemcpy(&(device_graph_frontier[root]), &true_flag, sizeof(bool), 
                hipMemcpyHostToDevice) );  // set root vertex as the first frontier
    cudaErrCheck( hipMemcpy(&(device_graph_visited[root]), &true_flag, sizeof(bool), 
                hipMemcpyHostToDevice) );  // set root vertex as visited
    cudaErrCheck( hipMemcpy(&(device_vpl[root]), &zero_flag, sizeof(uint32_t), 
                hipMemcpyHostToDevice) );  // set root vertex as visited


    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&h2d_copy_time, start_event, stop_event);

    
    // BFS traversal
    bool stop;
    hipEventRecord(start_event, 0);
   
    int k=0; 
    do
    {
        // Each iteration processes 
        //      one level of BFS traversal
        stop = false;
        cudaErrCheck( hipMemcpy(device_over, &stop, sizeof(bool), hipMemcpyHostToDevice) );

        // step 1
        BFS_kernel_1<<<num_block, num_thread_per_block>>>(d_graph, 
                device_graph_frontier, device_updating_graph_frontier, 
                device_graph_visited, device_vpl);

        // step 2
        BFS_kernel_2<<<num_block, num_thread_per_block>>>( 
                device_graph_frontier, device_updating_graph_frontier, 
                device_graph_visited,  
                device_over, vertex_cnt);

        cudaErrCheck( hipMemcpy(&stop, device_over, sizeof(bool), hipMemcpyDeviceToHost) );

        k++;
    }while(stop);

    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&kernel_time, start_event, stop_event);


    hipEventRecord(start_event, 0);

    cudaErrCheck( hipMemcpy(vproplist, device_vpl, vertex_cnt*sizeof(uint32_t), 
                hipMemcpyDeviceToHost) );
    
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&d2h_copy_time, start_event, stop_event);

    printf("== iteration #: %d\n", k);
#ifndef ENABLE_VERIFY
    printf("== host->device copy time: %f ms\n", h2d_copy_time);
    printf("== device->host copy time: %f ms\n", d2h_copy_time);
    printf("== kernel time: %f ms\n", kernel_time);
#endif
    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);

    // free graph struct on device side
    d_graph.cudaGraphFree();

    cudaErrCheck( hipFree(device_vpl) );

    cudaErrCheck( hipFree(device_graph_frontier) );
    cudaErrCheck( hipFree(device_updating_graph_frontier) );
    cudaErrCheck( hipFree(device_graph_visited) );
}

