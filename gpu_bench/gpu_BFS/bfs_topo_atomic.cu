#include "hip/hip_runtime.h"
//=================================================================//
// CUDA BFS kernel
// Topological-Driven: one node per thread, no frontier concept, 
//      use atomicMin for distance updates
// Reference: 
//      lonestar-GPU bfs_ls algorithm 
//=================================================================//
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>

#include "cudaGraph.h"

__global__ void initialize(uint32_t * d_graph_property, uint64_t num_vertex)
{
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if ( tid < num_vertex )
    {
        d_graph_property[tid] = MY_INFINITY;
    }
}

__device__
bool processnode(uint32_t * vplist, cudaGraph &graph, uint64_t vid) {
	if (vid >= graph.vertex_cnt) return false;
	bool changed = false;
	
    uint64_t e_begin = graph.get_firstedge_index(vid);
    uint64_t e_end = graph.get_edge_index_end(vid);
	for (unsigned ii = e_begin; ii < e_end; ++ii) 
    {
        uint64_t dest = graph.get_edge_dest(ii);
        uint32_t newlevel = vplist[vid]+1;
        if (newlevel < vplist[dest])
        {
            uint32_t oldlevel = atomicMin(&(vplist[dest]), newlevel);
            if (newlevel < oldlevel) changed = true; // the dest vertex is unvisited
            // else:  someone else already visited this vertex
        }
	}
	return changed;
}

__global__
void kernel(uint32_t * vplist, cudaGraph graph, bool *changed) {
	uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (processnode(vplist, graph, tid)) *changed = true;
}


void cuda_BFS(uint64_t * vertexlist, 
        uint64_t * edgelist, uint32_t * vproplist,
        uint64_t vertex_cnt, uint64_t edge_cnt,
        uint64_t root)
{
    uint32_t * device_vpl = 0;
    bool * device_over = 0;

    float h2d_copy_time = 0; // host to device data transfer time
    float d2h_copy_time = 0; // device to host data transfer time
    float kernel_time = 0;   // kernel execution time
    
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp,device);

    // Try to use as many threads as possible so that each thread
    //      is processing one vertex. If max thread is reached, 
    //      split them into multiple blocks.
    unsigned int num_thread_per_block = (unsigned int) vertex_cnt;
    if (num_thread_per_block > devProp.maxThreadsPerBlock)
        num_thread_per_block = devProp.maxThreadsPerBlock;
    unsigned int num_block = (unsigned int)ceil( vertex_cnt/(double)devProp.maxThreadsPerBlock );
    

    // malloc of gpu side
    cudaErrCheck( hipMalloc((void**)&device_vpl, vertex_cnt*sizeof(uint32_t)) );
    cudaErrCheck( hipMalloc((void**)&device_over, sizeof(bool)) );

    hipEvent_t start_event, stop_event;
    cudaErrCheck( hipEventCreate(&start_event) );
    cudaErrCheck( hipEventCreate(&stop_event) );
    
    // initialization
    initialize<<<num_block, num_thread_per_block>>>(device_vpl, vertex_cnt);
    
    // prepare graph struct
    //  one for host side, one for device side
    cudaGraph h_graph, d_graph;
    // here copy only the pointers
    h_graph.read(vertexlist, edgelist, vertex_cnt, edge_cnt);

    uint32_t zeronum=0;
    // memcpy from host to device
    hipEventRecord(start_event, 0);
   
    // copy graph data to device
    h_graph.cudaGraphCopy(&d_graph);

    cudaErrCheck( hipMemcpy(&(device_vpl[root]), &zeronum, sizeof(uint32_t), 
                hipMemcpyHostToDevice) );

    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&h2d_copy_time, start_event, stop_event);

    
    // BFS traversal
    bool stop;
    hipEventRecord(start_event, 0);
   
    int k=0; 
    do
    {
        // Each iteration processes 
        //      one level of BFS traversal
        stop = false;
        cudaErrCheck( hipMemcpy(device_over, &stop, sizeof(bool), hipMemcpyHostToDevice) );

        kernel<<<num_block, num_thread_per_block>>>(device_vpl, d_graph, device_over);

        cudaErrCheck( hipMemcpy(&stop, device_over, sizeof(bool), hipMemcpyDeviceToHost) );

        k++;
    }while(stop);

    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&kernel_time, start_event, stop_event);


    hipEventRecord(start_event, 0);

    cudaErrCheck( hipMemcpy(vproplist, device_vpl, vertex_cnt*sizeof(uint32_t), 
                hipMemcpyDeviceToHost) );
    
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&d2h_copy_time, start_event, stop_event);

    printf("== iteration #: %d\n", k);
#ifndef ENABLE_VERIFY
    printf("== host->device copy time: %f ms\n", h2d_copy_time);
    printf("== device->host copy time: %f ms\n", d2h_copy_time);
    printf("== kernel time: %f ms\n", kernel_time);
#endif
    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);

    // free graph struct on device side
    d_graph.cudaGraphFree();

    cudaErrCheck( hipFree(device_vpl) );
}

