#include "hip/hip_runtime.h"
//=================================================================//
// CUDA BFS kernel
// Data-Driven: data-driven algorithm, global worklist in memory
//      warp centric, one edge per thread, 
//      local thread aggregate tasks first before pushing to global worklist
//      need atomicAdd for maintaining the shared worklist
// Reference: 
// Rupesh Nasre, etc. Data-driven versus Topology-driven
//      Irregular Computations on GPUs
// Duane Merrill, etc. Scalable GPU Graph Traversal
//=================================================================//
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>

#include "cudaGraph.h"

#define WORKLIST_SIZE   16777216
#define LOCAL_SIZE      128

#define WARP_SZ     32
#define CHUNK_SZ    32

// a dummy worklist that you can only push or clear
typedef struct my_worklist
{
    void init(void)
    {
        cudaErrCheck( hipMalloc((void**)&item_array, WORKLIST_SIZE*sizeof(uint64_t)) );
        cudaErrCheck( hipMalloc((void**)&end, sizeof(uint32_t)) );
        clear();
    }

    void clear(void)
    {
        uint32_t zeronum=0;
        cudaErrCheck( hipMemcpy(end, &zeronum, sizeof(uint32_t), 
                hipMemcpyHostToDevice) );
    }

    void free(void)
    {
        cudaErrCheck( hipFree(item_array) );
        cudaErrCheck( hipFree(end) );
    }
    __device__ void pushRange(uint64_t * from_array, uint32_t num)
    {
        uint32_t old_end = atomicAdd(end, num);
        for (uint32_t i=0;i<num;i++)
        {
            item_array[i+old_end] = from_array[i];
        }
    }
    __device__ inline uint64_t get_item(unsigned index)
    {
        return item_array[index];
    }
    __device__ inline uint32_t get_item_num(void)
    {
            return (*end);
    }
    void host_initPush(uint64_t * from_array, uint32_t num)
    {
        cudaErrCheck( hipMemcpy(end, &num, sizeof(uint32_t), 
                hipMemcpyHostToDevice) );
        cudaErrCheck( hipMemcpy(item_array, from_array, num*sizeof(uint64_t), 
                hipMemcpyHostToDevice) );
    }

    uint64_t *item_array;
    uint32_t *end;
}my_worklist;

__global__ void initialize(uint32_t * d_graph_property, uint64_t num_vertex)
{
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if ( tid < num_vertex )
    {
        d_graph_property[tid] = MY_INFINITY;
    }
}

__global__
void kernel(uint32_t * vplist, cudaGraph graph, 
        my_worklist inworklist, my_worklist outworklist, unsigned curr_level) 
{
	uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t lane_id = tid % WARP_SZ;
    unsigned warp_id = tid / WARP_SZ;

    unsigned task_start = warp_id * CHUNK_SZ;
    unsigned task_end = task_start + CHUNK_SZ;

    if (task_start >= inworklist.get_item_num()) return;
    if (task_end > inworklist.get_item_num()) 
        task_end = inworklist.get_item_num();
    
    uint64_t local_worklist[LOCAL_SIZE]; 
    uint32_t work_size=0;
    
    for (unsigned id=task_start; id<task_end; id++)
    {
        uint64_t v = inworklist.get_item(id);
        uint64_t edge_ptr = graph.get_firstedge_index(v);
        uint64_t num_edge = graph.get_edge_index_end(v) - edge_ptr;

        for (uint64_t i=lane_id;i<num_edge;i+=WARP_SZ)
        {
            uint64_t vid = graph.get_edge_dest(i+edge_ptr);
            if (vplist[vid]==MY_INFINITY)
            {
                vplist[vid] = curr_level + 1;
                // push to local worklist
                local_worklist[work_size] = vid;
                work_size++;
                if (work_size==LOCAL_SIZE)
                {     
                    outworklist.pushRange(local_worklist, work_size);
                    work_size = 0;
                }
            }
        }

    }

    // push local worklist to shared worklist
    outworklist.pushRange(local_worklist, work_size);
}

void cuda_BFS(uint64_t * vertexlist, 
        uint64_t * edgelist, uint32_t * vproplist,
        uint64_t vertex_cnt, uint64_t edge_cnt,
        uint64_t root)
{
    uint32_t * device_vpl = 0;
    bool * device_over = 0;

    float h2d_copy_time = 0; // host to device data transfer time
    float d2h_copy_time = 0; // device to host data transfer time
    float kernel_time = 0;   // kernel execution time

    int device;
    hipGetDevice(&device);
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp,device);


    // Try to use as many threads as possible so that each thread
    //      is processing one vertex. If max thread is reached, 
    //      split them into multiple blocks.
    unsigned int num_thread_per_block = (unsigned int) vertex_cnt;
    if (num_thread_per_block > devProp.maxThreadsPerBlock)
        num_thread_per_block = devProp.maxThreadsPerBlock;
    unsigned int num_block = (unsigned int)ceil( vertex_cnt/(double)num_thread_per_block );

    // malloc of gpu side
    cudaErrCheck( hipMalloc((void**)&device_vpl, vertex_cnt*sizeof(uint32_t)) );
    cudaErrCheck( hipMalloc((void**)&device_over, sizeof(bool)) );

    hipEvent_t start_event, stop_event;
    cudaErrCheck( hipEventCreate(&start_event) );
    cudaErrCheck( hipEventCreate(&stop_event) );
    
    // initialization
    initialize<<<num_block, num_thread_per_block>>>(device_vpl, vertex_cnt);
    
    // prepare graph struct
    //  one for host side, one for device side
    cudaGraph h_graph, d_graph;
    // here copy only the pointers
    h_graph.read(vertexlist, edgelist, vertex_cnt, edge_cnt);

    // initialize the worklists for in & out
    my_worklist worklist1, worklist2;
    worklist1.init();
    worklist2.init();

    my_worklist * in_worklist = &worklist1;
    my_worklist * out_worklist = &worklist2;

    in_worklist->host_initPush(&root, 1);

    uint32_t zeronum=0;
    // memcpy from host to device
    hipEventRecord(start_event, 0);
   
    // copy graph data to device
    h_graph.cudaGraphCopy(&d_graph);

    cudaErrCheck( hipMemcpy(&(device_vpl[root]), &zeronum, sizeof(uint32_t), 
                hipMemcpyHostToDevice) );

    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&h2d_copy_time, start_event, stop_event);

    
    // BFS traversal
    hipEventRecord(start_event, 0);
   
    int curr=0;
    unsigned wl_size=1; 
    while(wl_size!=0)
    {
        // Each iteration processes 
        //      one level of BFS traversal

        num_thread_per_block = (unsigned int) wl_size;
        if (num_thread_per_block > devProp.maxThreadsPerBlock)
            num_thread_per_block = devProp.maxThreadsPerBlock;
        num_block = (unsigned int)ceil( wl_size/(double)num_thread_per_block );
        unsigned num_block_chunked = (unsigned int)ceil( num_block/(double)CHUNK_SZ )*WARP_SZ;


        kernel<<<num_block_chunked, num_thread_per_block>>>(
                device_vpl, d_graph, *in_worklist, *out_worklist, curr);
        
        my_worklist * temp=in_worklist;
        in_worklist = out_worklist;
        out_worklist = temp;
        cudaErrCheck( hipMemcpy(&wl_size, in_worklist->end, sizeof(uint32_t), hipMemcpyDeviceToHost) );
        out_worklist->clear();
        curr++;
    }

    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&kernel_time, start_event, stop_event);


    hipEventRecord(start_event, 0);

    cudaErrCheck( hipMemcpy(vproplist, device_vpl, vertex_cnt*sizeof(uint32_t), 
                hipMemcpyDeviceToHost) );
    
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&d2h_copy_time, start_event, stop_event);

    printf("== iteration #: %d\n", curr);
#ifndef ENABLE_VERIFY
    printf("== host->device copy time: %f ms\n", h2d_copy_time);
    printf("== device->host copy time: %f ms\n", d2h_copy_time);
    printf("== kernel time: %f ms\n", kernel_time);
#endif
    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);

    // free graph struct on device side
    d_graph.cudaGraphFree();

    in_worklist->free();
    out_worklist->free();
    cudaErrCheck( hipFree(device_vpl) );
}

