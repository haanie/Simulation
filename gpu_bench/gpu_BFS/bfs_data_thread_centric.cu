#include "hip/hip_runtime.h"
//=================================================================//
// CUDA BFS kernel
// Data-Driven: base data-driven algorithm, global worklist in memory
//      one vertex per thread (thread-centric), local thread aggregate 
//          tasks first before pushing to global worklist
//      perform vertex operations only on the worklist
//      need atomicAdd for maintaining the shared worklist
// Reference: 
// Rupesh Nasre, etc. Data-driven versus Topology-driven
//      Irregular Computations on GPUs
//=================================================================//
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>

#include "cudaGraph.h"

#define WORKLIST_SIZE   16777216
#define LOCAL_SIZE      128

// a dummy worklist that you can only push or clear
typedef struct my_worklist
{
    void init(void)
    {
        cudaErrCheck( hipMalloc((void**)&item_array, WORKLIST_SIZE*sizeof(uint64_t)) );
        cudaErrCheck( hipMalloc((void**)&end, sizeof(uint32_t)) );
        clear();
    }

    void clear(void)
    {
        uint32_t zeronum=0;
        cudaErrCheck( hipMemcpy(end, &zeronum, sizeof(uint32_t), 
                hipMemcpyHostToDevice) );
    }

    void free(void)
    {
        cudaErrCheck( hipFree(item_array) );
        cudaErrCheck( hipFree(end) );
    }
    __device__ void pushRange(uint64_t * from_array, uint32_t num)
    {
        uint32_t old_end = atomicAdd(end, num);
        for (uint32_t i=0;i<num;i++)
        {
            item_array[i+old_end] = from_array[i];
        }
    }
    __device__ inline uint64_t get_item(unsigned index)
    {
        return item_array[index];
    }
    __device__ inline uint32_t get_item_num(void)
    {
            return (*end);
    }
    void host_initPush(uint64_t * from_array, uint32_t num)
    {
        cudaErrCheck( hipMemcpy(end, &num, sizeof(uint32_t), 
                hipMemcpyHostToDevice) );
        cudaErrCheck( hipMemcpy(item_array, from_array, num*sizeof(uint64_t), 
                hipMemcpyHostToDevice) );
    }

    uint64_t *item_array;
    uint32_t *end;
}my_worklist;

__global__ void initialize(uint32_t * d_graph_property, uint64_t num_vertex)
{
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if ( tid < num_vertex )
    {
        d_graph_property[tid] = MY_INFINITY;
    }
}

__global__
void kernel(uint32_t * vplist, cudaGraph graph, 
        my_worklist inworklist, my_worklist outworklist) 
{
	uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= inworklist.get_item_num()) return;

    uint64_t v = inworklist.get_item(tid);
    uint64_t edge_begin = graph.get_firstedge_index(v);
    uint64_t edge_end = graph.get_edge_index_end(v);
    uint32_t curr = vplist[v];

    uint64_t local_worklist[LOCAL_SIZE]; 
    uint32_t work_size=0;
    for (uint64_t i=edge_begin;i<edge_end;i++)
    {
        uint64_t vid = graph.get_edge_dest(i);
        if (vplist[vid]==MY_INFINITY)
        {
            vplist[vid] = curr + 1;
            // push to local worklist
            local_worklist[work_size] = vid;
            work_size++;
            if (work_size==LOCAL_SIZE)
            {     
                outworklist.pushRange(local_worklist, work_size);
                work_size = 0;
            }
        }
    }
    // push local worklist to shared worklist
    outworklist.pushRange(local_worklist, work_size);
}

void cuda_BFS(uint64_t * vertexlist, 
        uint64_t * edgelist, uint32_t * vproplist,
        uint64_t vertex_cnt, uint64_t edge_cnt,
        uint64_t root)
{
    uint32_t * device_vpl = 0;
    bool * device_over = 0;

    float h2d_copy_time = 0; // host to device data transfer time
    float d2h_copy_time = 0; // device to host data transfer time
    float kernel_time = 0;   // kernel execution time

    int device;
    hipGetDevice(&device);
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp,device);


    // Try to use as many threads as possible so that each thread
    //      is processing one vertex. If max thread is reached, 
    //      split them into multiple blocks.
    unsigned int num_thread_per_block = (unsigned int) vertex_cnt;
    if (num_thread_per_block > devProp.maxThreadsPerBlock)
        num_thread_per_block = devProp.maxThreadsPerBlock;
    unsigned int num_block = (unsigned int)ceil( vertex_cnt/(double)num_thread_per_block );

    // malloc of gpu side
    cudaErrCheck( hipMalloc((void**)&device_vpl, vertex_cnt*sizeof(uint32_t)) );
    cudaErrCheck( hipMalloc((void**)&device_over, sizeof(bool)) );

    hipEvent_t start_event, stop_event;
    cudaErrCheck( hipEventCreate(&start_event) );
    cudaErrCheck( hipEventCreate(&stop_event) );
    
    // initialization
    initialize<<<num_block, num_thread_per_block>>>(device_vpl, vertex_cnt);
    
    // prepare graph struct
    //  one for host side, one for device side
    cudaGraph h_graph, d_graph;
    // here copy only the pointers
    h_graph.read(vertexlist, edgelist, vertex_cnt, edge_cnt);

    // initialize the worklists for in & out
    my_worklist worklist1, worklist2;
    worklist1.init();
    worklist2.init();

    my_worklist * in_worklist = &worklist1;
    my_worklist * out_worklist = &worklist2;

    in_worklist->host_initPush(&root, 1);

    uint32_t zeronum=0;
    // memcpy from host to device
    hipEventRecord(start_event, 0);
   
    // copy graph data to device
    h_graph.cudaGraphCopy(&d_graph);

    cudaErrCheck( hipMemcpy(&(device_vpl[root]), &zeronum, sizeof(uint32_t), 
                hipMemcpyHostToDevice) );

    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&h2d_copy_time, start_event, stop_event);

    
    // BFS traversal
    hipEventRecord(start_event, 0);
   
    int curr=0;
    unsigned wl_size=1; 
    while(wl_size!=0)
    {
        // Each iteration processes 
        //      one level of BFS traversal

        num_thread_per_block = (unsigned int) wl_size;
        if (num_thread_per_block > devProp.maxThreadsPerBlock)
            num_thread_per_block = devProp.maxThreadsPerBlock;
        num_block = (unsigned int)ceil( wl_size/(double)num_thread_per_block );


        kernel<<<num_block, num_thread_per_block>>>(device_vpl, d_graph, *in_worklist, *out_worklist);
        
        my_worklist * temp=in_worklist;
        in_worklist = out_worklist;
        out_worklist = temp;
        cudaErrCheck( hipMemcpy(&wl_size, in_worklist->end, sizeof(uint32_t), hipMemcpyDeviceToHost) );
        out_worklist->clear();
        curr++;
    }

    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&kernel_time, start_event, stop_event);


    hipEventRecord(start_event, 0);

    cudaErrCheck( hipMemcpy(vproplist, device_vpl, vertex_cnt*sizeof(uint32_t), 
                hipMemcpyDeviceToHost) );
    
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&d2h_copy_time, start_event, stop_event);

    printf("== iteration #: %d\n", curr);
#ifndef ENABLE_VERIFY
    printf("== host->device copy time: %f ms\n", h2d_copy_time);
    printf("== device->host copy time: %f ms\n", d2h_copy_time);
    printf("== kernel time: %f ms\n", kernel_time);
#endif
    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);

    // free graph struct on device side
    d_graph.cudaGraphFree();

    in_worklist->free();
    out_worklist->free();
    cudaErrCheck( hipFree(device_vpl) );
}

